#include "hip/hip_runtime.h"
//CUDA Headers
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
//C++ Headers
#include <iostream>
using namespace std;

__global__ void convolutionKernel(int * in_arr, int  * out_arr, int width, int height, float * stencil){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	int grid_width = gridDim.x * blockDim.x;
	int p = row * grid_width + col;

	if (row>0 && row <height - 1 && col>0 && col<width - 1){//Only pass stencil over pixels inside the border
		float ul = (float)(in_arr[p - width - 1]) * stencil[0];
		float um = (float)(in_arr[p - width]) * stencil[1];
		float ur = (float)(in_arr[p - width + 1]) * stencil[2];

		float ml = (float)(in_arr[p - 1]) * stencil[3];
		float mm = (float)(in_arr[p]) *stencil[4];
		float mr = (float)(in_arr[p + 1]) * stencil[5];

		float ll = (float)(in_arr[p + width - 1]) * stencil[6];
		float lm = (float)(in_arr[p + width]) * stencil[7];
		float lr = (float)(in_arr[p + width + 1]) * stencil[8];

		out_arr[p] = (int)(ul + um + ur + ml + mm + mr + ll + lm + lr);
	}
	else{//Border values should just be set to zero.
		out_arr[p] = 0;
	}
}

/*This kernel won't be implements until the convolutionKernel works.
__global__ void sobelKernel(int  * in_arr, int * out_arr, int const width, int const height, int * stencil){
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;

	if (row > 0 && row < height - 1 && col>0 && col < width - 1){//bad, need to find another way
		int p = row*width + col;//position

		int ul = (in_arr[p - width - 1]) * stencil[0][0];
		int um = (in_arr[p - width]) * stencil[0][1];
		int ur = (in_arr[p - width + 1]) * stencil[0][2];
		int ml = (in_arr[p - 1]) * stencil[1][0];
		int mm = (in_arr[p]) * stencil[1][1];
		int mr = (in_arr[p + 1]) * stencil[1][2];
		int ll = (in_arr[p + width - 1]) * stencil[2][0];
		int lm = (in_arr[p + width]) * stencil[2][1];
		int lr = (in_arr[p + width + 1]) * stencil[2][2];
		int x_sum = ul + um + ur + ml + mm + mr + ll + lm + lr;

		ul = (in_arr[p - width - 1]) * stencil[3][0];
		um = (in_arr[p - width]) * stencil[3][1];
		ur = (in_arr[p - width + 1]) * stencil[3][2];
		ml = (in_arr[p - 1]) * stencil[4][0];
		mm = (in_arr[p]) * stencil[4][1];
		mr = (in_arr[p + 1]) * stencil[4][2];
		ll = (in_arr[p + width - 1]) * stencil[5][0];
		lm = (in_arr[p + width]) * stencil[5][1];
		lr = (in_arr[p + width + 1]) * stencil[5][2];
		int y_sum = ul + um + ur + ml + mm + mr + ll + lm + lr;

		out_arr[p] = (int)pow((double)(y_sum*y_sum + x_sum*x_sum), 0.5);
	}
}*/

//If CUDA function was no succesful, print out the error and exit.
__host__ void HANDLE_CUDA_ERROR(hipError_t err, int line_num){
	if (err != hipSuccess){
		printf("\nCUDA ERROR in %s:\n%s at line %i.\n\n", __FILE__, hipGetErrorString(err), line_num);
		exit(EXIT_FAILURE);
	}
}

__host__ void launchKernel(int const  * in_arr, int  * out_arr, int width, int height, char const filter_type){
	//Check to see if there is a CUDA enabled device
	int count = 0;
	HANDLE_CUDA_ERROR(hipGetDeviceCount(&count), __LINE__);
	
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, 0);
	cout << "CUDA enabled device found: "<<props.name << " @" << props.clockRate/1000 << "Mhz" << endl;
	cout << "Applying Filter..." << endl;

	//Allocate device arrays
	int numElements = width * height;//width, height = orginal value + 2
	int * device_in_arr = 0;
	int * device_out_arr = 0;
	HANDLE_CUDA_ERROR(hipMalloc((void**)&device_in_arr, sizeof(int)*numElements), __LINE__);
	HANDLE_CUDA_ERROR(hipMalloc((void**)&device_out_arr, sizeof(int)*numElements), __LINE__);

	//Copy both arrays to device memory
	HANDLE_CUDA_ERROR(hipMemcpy(device_in_arr, in_arr, sizeof(int)*numElements, hipMemcpyHostToDevice), __LINE__);
	HANDLE_CUDA_ERROR(hipMemcpy(device_out_arr, out_arr, sizeof(int)*numElements, hipMemcpyHostToDevice), __LINE__);

	//Allocated device memeory for the the device stencil
	float * device_stencil = 0;
	HANDLE_CUDA_ERROR(hipMalloc((void**)&device_stencil, sizeof(float) * 9), __LINE__);

	int threadsPerBlock = 16;
	int blocksPerGrid = (int)ceil((double)numElements / (double)threadsPerBlock);//Guarantees that there are enough blocks in the grid for every element in the array.
	dim3 dimBlock(threadsPerBlock, threadsPerBlock); //16x16 thread blocks
	dim3 dimGrid(blocksPerGrid, blocksPerGrid);

	//Run the selected kernel
	if (filter_type == '8'){//CUDA Box Blur
		float boxblur_stencil[9] = { 1.f / 9 , 1.f / 9, 1.f / 9 , 1.f / 9, 1.f / 9, 1.f / 9 , 1.f / 9, 1.f / 9 , 1.f / 9 };
		HANDLE_CUDA_ERROR(hipMemcpy(device_stencil, boxblur_stencil, sizeof(float)*9, hipMemcpyHostToDevice), __LINE__);

		convolutionKernel << <dimGrid, dimBlock >> >(device_in_arr, device_out_arr, width, height, device_stencil);
		HANDLE_CUDA_ERROR(hipGetLastError(), __LINE__);

		HANDLE_CUDA_ERROR(hipFree(device_stencil), __LINE__);//Why is this giving an error?
	}
	else if (filter_type == '9'){//CUDA Sobel Operator
		int const sobel_stencil[6][3] = { { -1, 0, 1 }, { -2, 0, 2 }, { -1, 0, 1 },
		{ -1, -2, -1 }, { 0, 0, 0 }, { 1, 2, 1 } };
		//sobelKernel << <dimGrid, dimBlock >> >(device_in_arr, device_out_arr, width, height, sobel_stencil);
	}
	//Copy filtered array out of device and back to host
	HANDLE_CUDA_ERROR(hipMemcpy(out_arr, device_out_arr, sizeof(int)*numElements, hipMemcpyDeviceToHost), __LINE__);

	//Free allocated device arrays and reset the device.
	HANDLE_CUDA_ERROR(hipFree(device_in_arr), __LINE__);
	HANDLE_CUDA_ERROR(hipFree(device_out_arr), __LINE__);
	hipDeviceReset();

	cout << "Finished Applying Filter." << endl << endl;
}